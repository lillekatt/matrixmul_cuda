#include "hip/hip_runtime.h"
extern "C"{

  #include "matrixmul.h"

  #define TILE_WIDTH 32

  __global__ void MatrixMulKernel(MatrixMulOp *mat)
  {
      int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
      int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

      float Pvalue = 0;
      for(int k=0; k<mat->Width; ++k)
          Pvalue += mat->Md[Row * mat->Width + k] * mat->Nd[k * mat->Width + Col];

      mat->Pd[Row * mat->Width + Col] = Pvalue;
  }

  __global__ void MatrixMulKernelSh(MatrixMulOp *mat)
  {
      __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
      __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

      int bx = blockIdx.x;
      int by = blockIdx.y;
      int tx = threadIdx.x;
      int ty = threadIdx.y;

      int Row = by * TILE_WIDTH + ty;
      int Col = bx * TILE_WIDTH + tx;

      float Pvalue = 0;
      for(int m=0; m<mat->Width/TILE_WIDTH; ++m){
          Mds[ty][tx] = mat->Md[Row*mat->Width + (m*TILE_WIDTH + tx)];
          Nds[ty][tx] = mat->Nd[(m*TILE_WIDTH + ty)*mat->Width + Col];
          __syncthreads();

          for(int k=0; k<TILE_WIDTH; ++k)
              Pvalue += Mds[ty][k] * Nds[k][tx];
          __syncthreads();
      }

      mat->Pd[Row*mat->Width + Col] = Pvalue;
  }

}
